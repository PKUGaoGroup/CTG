#include "hip/hip_runtime.h"
/*
* Copyright 2023 Gao's lab, Peking University, CCME. All rights reserved.
*
* NOTICE TO LICENSEE:
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
* http://www.apache.org/licenses/LICENSE-2.0
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#include <stdio.h>
#include <string.h>
#include <vector>

__global__ void L1_Distance(const int n,const int i,const float *col_i,const float *matrix,float *dis_col_i)
{
	int j = blockIdx.x;
	if (j < n)
	{
		const float *col_j = &matrix[j*n];

		float temp_record = 0.f;
		for (int k = threadIdx.x; k < n; k = k + blockDim.x)
		{
			temp_record += fabsf(col_i[k] - col_j[k]);
		}
		atomicAdd(&dis_col_i[j], temp_record);
	}
}
int main(int argn, char *argv[])
{
	FILE *matrix_in = NULL;
	FILE *matrix_out = NULL;
	int n = 0;
	int iteration_numbers = 20;
	float alpha = 0.3f;
	hipblasHandle_t cuhandle;

	//input parameter
	for (int i = 0; i < argn; i = i + 1)
	{
		if (strcmp(argv[i], "-i") == 0)
		{
			i = i + 1;
			matrix_in = fopen(argv[i], "r");
		}
		else if (strcmp(argv[i], "-iteration_numbers") == 0)
		{
			i = i + 1;
			sscanf(argv[i], "%d", &iteration_numbers);
		}
		else if (strcmp(argv[i], "-alpha") == 0)
		{
			i = i + 1;
			sscanf(argv[i], "%f", &alpha);
		}
		else if (strcmp(argv[i], "-o") == 0)
		{
			i = i + 1;
			matrix_out = fopen(argv[i], "wb");
		}
	}

	//security check
	if (matrix_in == NULL)
	{
		printf("Please input a correct matrix name, after -i\n");
		getchar();
		return 0;
	}
	if (iteration_numbers <= 0)
	{
		printf("Please make sure iteration numbers > 0!\n");
		getchar();
		return 0;
	}

	//read matrix(start from 1)
	std::vector<int> array_i;
	std::vector<int> array_j;
	std::vector<float> array_value;
	int non_zero_count=0;
	while (true)
	{
		int i, j;
		float value;
		int pan = fscanf(matrix_in, "%d %d %f", &i, &j, &value);
		if (pan == EOF)
		{
			break;
		}
		i = i - 1;
		j = j - 1;
		array_i.push_back(i);
		array_j.push_back(j);
		array_value.push_back(value);
		if (i > n)
		{
			n = i;
		}
		if (j > n)
		{
			n = j;
		}
		non_zero_count+=1;
	}
	n = n + 1;
	printf("Matrix size is %d\nNonzero number %d\n", n,non_zero_count);

	//initial GPU and copy data to GPU
	float *h_origin_matrix = NULL;
	h_origin_matrix = (float*)malloc(sizeof(float)*n*n);
	memset(h_origin_matrix,0,sizeof(float)*n*n);
	float *d_origin_matrix = NULL;
	hipMalloc((void**)&d_origin_matrix, sizeof(float)*n*n);
	for (int k = 0; k < array_i.size(); k = k + 1)
	{
		int i = array_i[k];
		int j = array_j[k];
		float value = array_value[k];
		size_t serial = (size_t)i * n + j;
		h_origin_matrix[serial] = value;
		serial = (size_t)i + n * j;
		h_origin_matrix[serial] = value;
		if (i == j)
		{
			h_origin_matrix[serial] = 0.f;
		}//diagonal element set to 0.f
	}
	hipError_t cudaerror=hipMemcpy(d_origin_matrix, h_origin_matrix, sizeof(float)*n*n, hipMemcpyHostToDevice);
	
	//(matrix normalization)
	std::vector<int>col_zero_record;
	hipblasCreate(&cuhandle);
	for (size_t col_i = 0; col_i < n; col_i = col_i + 1)
	{
		float sum;
		hipblasSasum(cuhandle, n, &d_origin_matrix[col_i*n], 1, &sum);
		if (fabsf(sum) < 1.e-6f)
		{
			col_zero_record.push_back((int)col_i);
			hipMemset(&d_origin_matrix[col_i*n], 0, sizeof(float)*n);
		}
		else
		{
			sum = 1.f / sum;
			hipblasSscal(cuhandle, n, &sum, &d_origin_matrix[col_i*n], 1);
		}
	}

	//itetration (matrix multiplication)
	const float one = 1.f;
	const float zero = 0.f;
	float *d_sum_matrix = NULL;
	float *d_k_matrix = NULL;
	float *d_k_matrix_copy = NULL;
	hipMalloc((void**)&d_sum_matrix, sizeof(float)*n*n);
	hipMemset(d_sum_matrix, 0, sizeof(float)*n*n);
	hipMalloc((void**)&d_k_matrix, sizeof(float)*n*n);
	hipMemcpy(d_k_matrix, d_origin_matrix, sizeof(float)*n*n, hipMemcpyDeviceToDevice);
	hipMalloc((void**)&d_k_matrix_copy, sizeof(float)*n*n);
	hipMemcpy(d_k_matrix_copy, d_origin_matrix, sizeof(float)*n*n, hipMemcpyDeviceToDevice);
	printf("iteration start!\n");
	for (int iteration_i = 0; iteration_i < iteration_numbers; iteration_i = iteration_i + 1)
	{
		printf("iteration %d!\n",iteration_i);
		float factor = expf(-alpha*(iteration_i + 1));
		hipblasSaxpy(cuhandle, n*n, &factor, d_k_matrix, 1, d_sum_matrix, 1);
		hipblasSgemm(cuhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_k_matrix, n, d_origin_matrix, n, &zero, d_k_matrix_copy, n);
		hipMemcpy(d_k_matrix, d_k_matrix_copy, sizeof(float)*n*n, hipMemcpyDeviceToDevice);
	}
	printf("iteration done!\n");

	//(calculate L1 distance)
	printf("L1 distance start!\n");
	float *d_col_i = NULL;
	hipMalloc((void**)&d_col_i, sizeof(float)*n);
	float *h_L1_matrix = NULL;
	h_L1_matrix = (float*)malloc(sizeof(float)*n*n);
	int current_zero_id = 0;
	for (int k = 0; k < n; k = k + 1)
	{
		if (current_zero_id < col_zero_record.size())
		{
			if (col_zero_record[current_zero_id] == k)
			{
				current_zero_id += 1;
				continue;
			}
		}
		hipMemset(d_col_i, 0, sizeof(float)*n);
		L1_Distance << <n, 256 >> >
			(n, k, &d_sum_matrix[k * n], d_sum_matrix, d_col_i);
		hipMemcpy(&h_L1_matrix[k * n], d_col_i, sizeof(float)*n, hipMemcpyDeviceToHost);
		if(k%1000==0)
		{
			printf("L1 distance %d\n",k);
		}
	}
	for (int i = 0; i < col_zero_record.size(); i = i + 1)
	{
		for (int j = 0; j < n; j = j + 1)
		{
			h_L1_matrix[col_zero_record[i] * n + j] = 0.f;
			h_L1_matrix[col_zero_record[i] + n * j] = 0.f;
		}
	}
	printf("L1 distance done!\n");
	

	if (matrix_out == NULL)
	{
		matrix_out = fopen("Distance_Matrix.dat", "wb");
	}
	fwrite(h_L1_matrix, sizeof(float), n*n, matrix_out);
	fclose(matrix_out);

	printf("done!");
	return 0;
}